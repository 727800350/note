
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float *d_out, float *d_in){
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f;
}

int main(int argc, char **argv){
	const int array_size = 64;
	const int array_bytes = array_size * sizeof(float);

	// generate the input data on the host
	float h_in[array_size];
	for(int i = 0; i < array_size; i++){
		h_in[i] = float(i);
	}
	float h_out[array_size];

	// declare gpu memory
	float *d_in;
	float *d_out;
	
	// allocate gpu memory
	hipMalloc((void **)&d_in, array_bytes);
	hipMalloc((void **)&d_out, array_bytes);

	// transfer the array to the gpu
	hipMemcpy(d_in, h_in, array_bytes, hipMemcpyHostToDevice);
	
	// launch the kernel
	square<<<1, array_size>>>(d_out, d_in);

	// copy back the result to cpu
	hipMemcpy(h_out, d_out, array_bytes, hipMemcpyDeviceToHost);

	for(int i = 0; i < array_size; i++){
		fprintf(stdout, "h_out[i]\n");
	}

	// free gpu memory allocation
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
